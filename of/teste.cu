#include "hip/hip_runtime.h"
#include "STM.cuh"
//#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CHECK_ERROR(func, msg) ({ \
	hipError_t hipError_t; \
	if (hipSuccess != (hipError_t = func)) { \
		fprintf(stderr, #func ": in " __FILE__ ":%i : " msg "\n   > %s\n", \
		__LINE__, hipGetErrorString(hipError_t)); \
    *((int*)0x0) = 0; /* exit(-1); */ \
	} \
  hipError_t; \
})

#define N_OBJECTS 100

__device__ float rand_() {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
       hiprandState state;
        hiprand_init(clock64(), i, 0, &state);

       return hiprand_uniform(&state);

}

__global__
void foo(STMData* stm_data){
   printf("aqui\n.");
   int id = threadIdx.x + blockIdx.x * blockDim.x;
   TX_Data* tx_data = TX_Init(stm_data,id);
   int n_trans = 10;
   int trans = 0;
   while(trans<n_trans)
   {
        int o1 = (int)rand_() % (N_OBJECTS -1);
        int o2;
        do{
            o2 = (int) rand_() % (N_OBJECTS -1);
        }while(o1 == o2);
      printf("o1: %d, o2: %d\n",o1,o2);

   
        int aborted;
        do{
            TX_Start(stm_data,tx_data);
            aborted = 0;
            int* ptr1 = TX_Open_Write(stm_data,tx_data,o1);
            if(stm_data->tr_state[tx_data->tr_id] != ABORTED)
            {
                int* ptr2 = TX_Open_Write(stm_data,tx_data,o2);
                if(ptr2 !=0 )
                {
                    if(*ptr1 > 10)
                    {
                    *ptr1 -= 10;
                    *ptr2 += 10;
                    }
                    TX_commit(stm_data,tx_data);
                    if(stm_data->tr_state[tx_data->tr_id] == COMMITTED)
                        {trans ++;
                          __syncthreads();
                          TX_garbage_collect(stm_data,tx_data);
                          __syncthreads();
                        }
                }
            }
            assert(stm_data->tr_state[tx_data->tr_id] != ACTIVE);
            if(stm_data->tr_state[tx_data->tr_id] == ABORTED)
            {
                TX_abort_tr(stm_data,tx_data);
                aborted = 1;
            }
            if(!aborted)
                assert(stm_data->tr_state[tx_data->tr_id] != ACTIVE);
      
        }while(aborted);
   }
 //   return NULL;
    
}

int main()
{
  int num_objects = N_OBJECTS;
  int num_locators = MAX_LOCATORS;
  int num_tx = 1;

  int num_blocks = num_tx;
  int num_threads = 1;
 
  STMData* stm_data = STM_start(num_objects, num_tx, num_locators); 
  init_objects(stm_data,num_objects,100);
  init_locators(stm_data,num_tx,num_locators);
  STMData *d_stm_data = STM_copy_to_device(stm_data);

  foo<<<num_blocks,num_threads>>>(d_stm_data);
  hipError_t kernelErr = hipGetLastError();
  if(kernelErr != hipSuccess) printf("Error kernel: %s\n", hipGetErrorString(kernelErr));

  CUDA_CHECK_ERROR( hipDeviceSynchronize() , " synchronize ");
  //kernelErr = hipGetLastError();
  //if(kernelErr != hipSuccess) printf("Error synchronize: %s\n", hipGetErrorString(kernelErr));
  
  STM_copy_from_device(d_stm_data,stm_data);
  printf("FIM!\n");
  print_stats(stm_data);
  
  
}

